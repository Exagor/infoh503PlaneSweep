#include "hip/hip_runtime.h"
#include "main.cuh"

#include <cstdio>

//#define DEBUG
#define BLOCKSIZE 16 //16 or 32
__constant__ float d_param_ref[21];
__constant__ float d_param_cam[54]; //18*3=54

// Those functions are an example on how to call cuda functions from the main.cpp
__global__ void dev_test_vecAdd(int* A, int* B, int* C, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= N) return;

	C[i] = A[i] + B[i];
}

void wrap_test_vectorAdd() {
	printf("Testing GPU with vector addition:\n");

	int N = 3;
	int a[] = { 1, 2, 3 };
	int b[] = { 1, 2, 3 };
	int c[] = { 0, 0, 0 };

	int* dev_a, * dev_b, * dev_c;

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	hipMemcpy(dev_a, a, N * sizeof(int),
		hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int),
		hipMemcpyHostToDevice);

	dev_test_vecAdd <<<1, N>>> (dev_a, dev_b, dev_c, N);

	hipMemcpy(c, dev_c, N * sizeof(int),
		hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	printf("%s\n", hipGetErrorString(hipGetLastError()));
	
	for (int i = 0; i < N; ++i) {
		printf("%i + %i = %i\n", a[i], b[i], c[i]);
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
}

//Functions to time the kernel
hipEvent_t start_cuda_timer()
{
	hipEvent_t start;
	hipEventCreate(&start);
	hipEventRecord(start, NULL);
	return start;
}

void end_cuda_timer(hipEvent_t start, const char* name)
{
	hipEvent_t stop;
	hipEventCreate(&stop);
	hipEventRecord(stop, NULL);
	hipEventSynchronize(stop);
	float millisec;
	hipEventElapsedTime(&millisec, start, stop);

	//printf("%s:\n", name);
	printf("Time for kernel %s: %f ms\n",name, millisec);
}

float* get_params_cam(const cam camera, const int is_ref, std::vector<cam> const& cam_vector) {
	if (is_ref) {
		// Copy the camera parameters into the array
		float* params = (float*)malloc(sizeof(float) * 21);
		int index = 0;
		// K_inv
		for (int i = 0; i < 9; ++i) {
			params[index++] = (float)camera.p.K_inv[i];
		}
		// R_inv
		for (int i = 0; i < 9; ++i) {
			params[index++] = (float)camera.p.R_inv[i];
		}
		// t_inv
		for (int i = 0; i < 3; ++i) {
			params[index++] = (float)camera.p.t_inv[i];
		}
		return params;
	}
	else {
		float* params = (float*)malloc(sizeof(float) * 18 * 3);
		int index = 0;
		for (auto& cam : cam_vector)
		{
			if (cam.name == camera.name)
				continue;
			// R
			for (int i = 0; i < 9; ++i) {
				params[index++] = (float)cam.p.R[i];
			}
			// t
			for (int i = 0; i < 3; ++i) {
				params[index++] = (float)cam.p.t[i];
			}
			// K
			for (int i = 0; i < 6; ++i) {
				params[index++] = (float)cam.p.K[i];
			}
		}
		return params;
	}
}

__global__ void naive_sweeping_plane_kernel(
	const uint8_t* im_ref,
	const uint8_t* im_cam,
	const double* param_ref,
	const double* param_cam,
	__half* cost_volume, //was float
	const unsigned int width, const unsigned int height,
	int z_planes,
	int window)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int zi = blockIdx.z;

	if (x >= width || y >= height || zi >= z_planes) //handle threads/blocks out of bounds
		return;
	// (1) compute the projection index
	//double x_proj, y_proj; //At the end see if it optimize the code by putting float instead of double
	float z = 0.3f * 1.1f / (0.3f + ((float)zi / z_planes) * (1.1f - 0.3f)); //Defined in constants.hpp

	// 2D ref camera point to 3D in ref camera coordinates (p * K_inv)
	float X_ref = (param_ref[0] * x + param_ref[1] * y + param_ref[2]) * z; //Was float type
	float Y_ref = (param_ref[3] * x + param_ref[4] * y + param_ref[5]) * z;
	float Z_ref = (param_ref[6] * x + param_ref[7] * y + param_ref[8]) * z;

	// 3D in ref camera coordinates to 3D world
	float X = param_ref[9] * X_ref + param_ref[10] * Y_ref + param_ref[11] * Z_ref - param_ref[18];
	float Y = param_ref[12] * X_ref + param_ref[13] * Y_ref + param_ref[14] * Z_ref - param_ref[19];
	float Z = param_ref[15] * X_ref + param_ref[16] * Y_ref + param_ref[17] * Z_ref - param_ref[20];

	// 3D world to projected camera 3D coordinates
	float X_proj = param_cam[21] * X + param_cam[22] * Y + param_cam[23] * Z - param_cam[30];
	float Y_proj = param_cam[24] * X + param_cam[25] * Y + param_cam[26] * Z - param_cam[31];
	float Z_proj = param_cam[27] * X + param_cam[28] * Y + param_cam[29] * Z - param_cam[32];

	// Projected camera 3D coordinates to projected camera 2D coordinates
	float x_proj = (param_cam[33] * X_proj / Z_proj + param_cam[34] * Y_proj / Z_proj + param_cam[35]);
	float y_proj = (param_cam[36] * X_proj / Z_proj + param_cam[37] * Y_proj / Z_proj + param_cam[38]);
	//float z_proj = Z_proj;

	// Verification it's not out of bounds
	x_proj = x_proj < 0 || x_proj >= width ? 0 : roundf(x_proj);
	y_proj = y_proj < 0 || y_proj >= height ? 0 : roundf(y_proj);

	// (2) Compute the SAD between the windows of ref and cam
	//float cost = compute_cost(im_ref, im_cam, width, height, x_proj, y_proj, x, y, window);
	float cost = 0.0f;
	int half = window / 2;
	float count = 0;

	for (int dy = -half; dy <= half; dy++) {
		for (int dx = -half; dx <= half; dx++) {
			int rx = x + dx;
			int ry = y + dy; //Could be outside this for loop, but it would be less readable
			int px = (int)roundf(x_proj) + dx;
			int py = (int)roundf(y_proj) + dy;

			if (rx >= 0 && ry >= 0 && rx < width && ry < height &&
				px >= 0 && py >= 0 && px < width && py < height) {

				int ref_idx = INDEX_2D(ry, rx, width);//ry * width + rx;
				int cam_idx = INDEX_2D(py, px, width);//py * height + px; why was it height here?
				cost += fabsf((float)(im_ref[ref_idx]) - (float)(im_cam[cam_idx])); //was float
				count += 1.0f; //Was 1.0f
			}
		}
	}
	if (count > 0) {
		cost = cost / count;
	}
	else {
		cost = 255.0f; //If no pixels were counted, return a high cost
	}
	// (3) Store the min cost in the cost volume
	cost_volume[INDEX_3D(zi, y, x, height, width)] = fminf(cost_volume[INDEX_3D(zi, y, x, height, width)],__float2half(cost));
}

__global__ void params_sweeping_plane_kernel(
	const uint8_t* im_ref,
	const uint8_t* im_cam,
	__half* cost_volume,
	const int cam_nb,
	const unsigned int width, const unsigned int height,
	int z_planes,
	int window)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int zi = blockIdx.z;

	if (x >= width || y >= height || zi >= z_planes) //handle threads/blocks out of bounds
		return;
	// (1) compute the projection index
	int cam_index = (cam_nb * 18); //18*3=54
	float z = 0.3f * 1.1f / (0.3f + ((float)zi / z_planes) * (1.1f - 0.3f)); //Defined in constants.hpp

	// 2D ref camera point to 3D in ref camera coordinates (p * K_inv)
	float X_ref = (d_param_ref[0] * x + d_param_ref[1] * y + d_param_ref[2]) * z;
	float Y_ref = (d_param_ref[3] * x + d_param_ref[4] * y + d_param_ref[5]) * z;
	float Z_ref = (d_param_ref[6] * x + d_param_ref[7] * y + d_param_ref[8]) * z;

	// 3D in ref camera coordinates to 3D world
	float X = d_param_ref[9] * X_ref + d_param_ref[10] * Y_ref + d_param_ref[11] * Z_ref - d_param_ref[18];
	float Y = d_param_ref[12] * X_ref + d_param_ref[13] * Y_ref + d_param_ref[14] * Z_ref - d_param_ref[19];
	float Z = d_param_ref[15] * X_ref + d_param_ref[16] * Y_ref + d_param_ref[17] * Z_ref - d_param_ref[20];

	// 3D world to projected camera 3D coordinates
	float X_proj = d_param_cam[0 + cam_index] * X + d_param_cam[1 + cam_index] * Y + d_param_cam[2 + cam_index] * Z - d_param_cam[9 + cam_index];
	float Y_proj = d_param_cam[3 + cam_index] * X + d_param_cam[4 + cam_index] * Y + d_param_cam[5 + cam_index] * Z - d_param_cam[10 + cam_index];
	float Z_proj = d_param_cam[6 + cam_index] * X + d_param_cam[7 + cam_index] * Y + d_param_cam[8 + cam_index] * Z - d_param_cam[11 + cam_index];

	// Projected camera 3D coordinates to projected camera 2D coordinates
	float x_proj = (d_param_cam[12 + cam_index] * X_proj / Z_proj + d_param_cam[13 + cam_index] * Y_proj / Z_proj + d_param_cam[14 + cam_index]);
	float y_proj = (d_param_cam[15 + cam_index] * X_proj / Z_proj + d_param_cam[16 + cam_index] * Y_proj / Z_proj + d_param_cam[17 + cam_index]);
	//float z_proj = Z_proj;

	// Verification it's not out of bounds
	x_proj = x_proj < 0 || x_proj >= width ? 0 : roundf(x_proj);
	y_proj = y_proj < 0 || y_proj >= height ? 0 : roundf(y_proj);

	// (2) Compute the SAD between the windows of ref and cam
	float cost = 0.0f;
	const int half = window / 2;
	float count = 0;

	int px_base = (int)x_proj; //already cast outside the loop
	int py_base = (int)y_proj;
	for (int dy = -half; dy <= half; dy++) {
		for (int dx = -half; dx <= half; dx++) {
			int rx = x + dx;
			int ry = y + dy;
			int px = px_base + dx;
			int py = py_base + dy;

			if (rx < 0 || ry < 0 || rx >= width || ry >= height) continue;
			if (px < 0 || py < 0 || px >= width || py >= height) continue;

			int ref_idx = INDEX_2D(ry, rx, width);
			int cam_idx = INDEX_2D(py, px, width);
			cost += fabsf((float)(im_ref[ref_idx]) - (float)(im_cam[cam_idx]));
			count += 1.0f;
		}
	}
	if (count > 0) {
		cost = cost / count;
	}
	else {
		cost = 255.0f; //If no pixels were counted, return a high cost
	}
	// (3) Store the min cost in the cost volume
	int idx = INDEX_3D(zi, y, x, height, width);
	cost_volume[idx] = fminf(cost_volume[idx], __float2half(cost));
}

__global__ void shared_sweeping_plane_kernel(
	const uint8_t* im_ref,
	const uint8_t* im_cam,
	__half* cost_volume,
	const int cam_nb,
	const unsigned int width, const unsigned int height,
	const int z_planes,
	const int window)
{
	extern __shared__ uint8_t shared_ref[];

	const int pad = window / 2; //For the borders of the block
	const int shared_width = blockDim.x + 2 * pad;
	const int shared_height = blockDim.y + 2 * pad;

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	const int x = blockIdx.x * blockDim.x + tx;
	const int y = blockIdx.y * blockDim.y + ty;
	const int zi = blockIdx.z;

	if (x >= width || y >= height || zi >= z_planes) return;

	// Load im_ref into shared memory (with borders)
	const int lx = tx + pad;
	const int ly = ty + pad;

	// Fill center
	shared_ref[INDEX_2D(ly, lx, shared_width)] = im_ref[INDEX_2D(y, x, width)];

	// Borders - left/right
	if (tx < pad) {
		int left_x = x - pad;
		shared_ref[INDEX_2D(ly, tx, shared_width)] = (left_x >= 0) ? im_ref[INDEX_2D(y, left_x, width)] : 0; //fill border with 0

		int right_x = x + blockDim.x;
		shared_ref[INDEX_2D(ly, tx + blockDim.x + pad, shared_width)] = (right_x < width) ? im_ref[INDEX_2D(y,right_x, width)] : 0;
	}

	// Borders - top/bottom
	if (ty < pad) {
		int top_y = y - pad;
		shared_ref[INDEX_2D(ty, lx, shared_width)] = (top_y >= 0) ? im_ref[INDEX_2D(top_y, x, width)] : 0;

		int bottom_y = y + blockDim.y;
		shared_ref[INDEX_2D(ty + blockDim.y + pad, lx, shared_width)] = (bottom_y < height) ? im_ref[INDEX_2D(bottom_y, x, width)] : 0;
	}

	// Corners
	if (tx < pad && ty < pad) {
		int tl_x = x - pad, tl_y = y - pad;
		int tr_x = x + blockDim.x, tr_y = y - pad;
		int bl_x = x - pad, bl_y = y + blockDim.y;
		int br_x = x + blockDim.x, br_y = y + blockDim.y;

		shared_ref[INDEX_2D(ty, tx, shared_width)] =
			(tl_x >= 0 && tl_y >= 0) ? im_ref[INDEX_2D(tl_y, tl_x, width)] : 0;

		shared_ref[INDEX_2D(ty, tx + blockDim.x + pad, shared_width)] =
			(tr_x < width && tr_y >= 0) ? im_ref[INDEX_2D(tr_y, tr_x, width)] : 0;

		shared_ref[INDEX_2D(ty + blockDim.y + pad, tx, shared_width)] =
			(bl_x >= 0 && bl_y < height) ? im_ref[INDEX_2D(bl_y, bl_x, width)] : 0;

		shared_ref[INDEX_2D(ty + blockDim.y + pad, tx + blockDim.x + pad, shared_width)] =
			(br_x < width && br_y < height) ? im_ref[INDEX_2D(br_y, br_x, width)] : 0;
	}

	__syncthreads();

	// (1) compute the projection index
	int cam_index = (cam_nb * 18); //18*3=54
	float z = 0.3f * 1.1f / (0.3f + ((float)zi / z_planes) * (1.1f - 0.3f)); //Defined in constants.hpp

	// 2D ref camera point to 3D in ref camera coordinates (p * K_inv)
	float X_ref = (d_param_ref[0] * x + d_param_ref[1] * y + d_param_ref[2]) * z;
	float Y_ref = (d_param_ref[3] * x + d_param_ref[4] * y + d_param_ref[5]) * z;
	float Z_ref = (d_param_ref[6] * x + d_param_ref[7] * y + d_param_ref[8]) * z;

	// 3D in ref camera coordinates to 3D world
	float X = d_param_ref[9] * X_ref + d_param_ref[10] * Y_ref + d_param_ref[11] * Z_ref - d_param_ref[18];
	float Y = d_param_ref[12] * X_ref + d_param_ref[13] * Y_ref + d_param_ref[14] * Z_ref - d_param_ref[19];
	float Z = d_param_ref[15] * X_ref + d_param_ref[16] * Y_ref + d_param_ref[17] * Z_ref - d_param_ref[20];

	// 3D world to projected camera 3D coordinates
	float X_proj = d_param_cam[0 + cam_index] * X + d_param_cam[1 + cam_index] * Y + d_param_cam[2 + cam_index] * Z - d_param_cam[9 + cam_index];
	float Y_proj = d_param_cam[3 + cam_index] * X + d_param_cam[4 + cam_index] * Y + d_param_cam[5 + cam_index] * Z - d_param_cam[10 + cam_index];
	float Z_proj = d_param_cam[6 + cam_index] * X + d_param_cam[7 + cam_index] * Y + d_param_cam[8 + cam_index] * Z - d_param_cam[11 + cam_index];

	// Projected camera 3D coordinates to projected camera 2D coordinates
	float x_proj = (d_param_cam[12 + cam_index] * X_proj / Z_proj + d_param_cam[13 + cam_index] * Y_proj / Z_proj + d_param_cam[14 + cam_index]);
	float y_proj = (d_param_cam[15 + cam_index] * X_proj / Z_proj + d_param_cam[16 + cam_index] * Y_proj / Z_proj + d_param_cam[17 + cam_index]);

	// Verification it's not out of bounds
	x_proj = x_proj < 0 || x_proj >= width ? 0 : roundf(x_proj);
	y_proj = y_proj < 0 || y_proj >= height ? 0 : roundf(y_proj);

	// (2) Compute the SAD between the windows of ref and cam
	float cost = 0.0f;
	float count = 0;

	int px_base = (int)x_proj;
	int py_base = (int)y_proj;
	for (int dy = -pad; dy <= pad; dy++) { //pad is half the window size
		for (int dx = -pad; dx <= pad; dx++) {
			int rx = lx + dx;
			int ry = ly + dy;

			int px = px_base + dx;
			int py = py_base + dy;

			//if (rx < 0 || ry < 0 || rx >= width || ry >= height) continue; //Don't need to verify because lx = tx + pad
			if (px < 0 || py < 0 || px >= width || py >= height) continue;

			int cam_idx = INDEX_2D(py, px, width);
			float ref_val = (float)shared_ref[INDEX_2D(ry, rx, shared_width)];
			float cam_val = (float)im_cam[cam_idx];
			cost += fabsf(ref_val - cam_val);
			count += 1.0f;
		}
	}
	cost = (count > 0) ? cost / count : 255.0f;

	// (3) Store the min cost in the cost volume
	int idx = INDEX_3D(zi, y, x, height, width);
	cost_volume[idx] = fminf(cost_volume[idx], __float2half(cost));
}

void wrap_plane_sweep(cam const ref, std::vector<cam> const &cam_vector, int z_planes, int window, __half* h_cost_volume)
{
	hipEvent_t start;
	const unsigned int height = ref.height;
	const unsigned int width = ref.width;
	const unsigned int img_size = width * height;
	const unsigned int volume_size = img_size * z_planes;

	uint8_t* d_im_ref = 0;
	uint8_t* d_im_cam1 = 0;
	uint8_t* d_im_cam2 = 0;
	uint8_t* d_im_cam3 = 0;
	__half* d_cost_volume = 0; //previously float
	//For the params used in naive
	//double* d_param_ref = 0;
	//double* d_param_cam1 = 0;
	//double* d_param_cam2 = 0;
	//double* d_param_cam3 = 0;

	CHK(hipSetDevice(0));

	// init variables to contain image
	uint8_t* ref_flattened = new uint8_t[img_size];
	uint8_t* cam1_flattened = new uint8_t[img_size];
	uint8_t* cam2_flattened = new uint8_t[img_size];
	uint8_t* cam3_flattened = new uint8_t[img_size];
	// flatten the matrix
	for (int y = 0; y < ref.height; ++y) {
		for (int x = 0; x < ref.width; ++x) {
			ref_flattened[INDEX_2D(y,x,ref.width)] = ref.YUV[0].at<uint8_t>(y, x);
		}
	}
	for (int y = 0; y < cam_vector.at(1).height; ++y) {
		for (int x = 0; x < cam_vector.at(1).width; ++x) {
			cam1_flattened[INDEX_2D(y, x, cam_vector.at(1).width)] = cam_vector.at(1).YUV[0].at<uint8_t>(y, x);
		}
	}
	for (int y = 0; y < cam_vector.at(2).height; ++y) {
		for (int x = 0; x < cam_vector.at(2).width; ++x) {
			cam2_flattened[INDEX_2D(y, x, cam_vector.at(2).width)] = cam_vector.at(2).YUV[0].at<uint8_t>(y, x);
		}
	}
	for (int y = 0; y < cam_vector.at(3).height; ++y) {
		for (int x = 0; x < cam_vector.at(3).width; ++x) {
			cam3_flattened[INDEX_2D(y, x, cam_vector.at(3).width)] = cam_vector.at(3).YUV[0].at<uint8_t>(y, x);
		}
	}
	
#ifdef DEBUG // print the flattened matrix
	for (int y = 0; y < ref.height/10; ++y) {
		for (int x = 0; x < ref.width/10; ++x) {
			printf("%d ", ref_flattened[INDEX_2D(y, x, ref.width)]);
		}
		printf("\n");
	}
	for (int y = 0; y < cam_vector.at(1).height/10; ++y) {
		for (int x = 0; x < cam_vector.at(1).width/10; ++x) {
			printf("%d ", cam_flattened[INDEX_2D(y, x, cam_vector.at(1).width)]);
		}
		printf("\n");
	}
#endif

	CHK(hipMalloc((void**) &d_im_ref, img_size * sizeof(uint8_t)));
	CHK(hipMalloc((void**) &d_im_cam1, img_size * sizeof(uint8_t)));
	CHK(hipMalloc((void**) &d_im_cam2, img_size * sizeof(uint8_t)));
	CHK(hipMalloc((void**) &d_im_cam3, img_size * sizeof(uint8_t)));
	CHK(hipMalloc((void**) &d_cost_volume, volume_size * sizeof(__half)));//previously float
	CHK(hipMemcpy(d_im_ref, ref_flattened, img_size * sizeof(uint8_t), hipMemcpyHostToDevice));
	CHK(hipMemcpy(d_im_cam1, cam1_flattened, img_size * sizeof(uint8_t), hipMemcpyHostToDevice));
	CHK(hipMemcpy(d_im_cam2, cam2_flattened, img_size * sizeof(uint8_t), hipMemcpyHostToDevice));
	CHK(hipMemcpy(d_im_cam3, cam3_flattened, img_size * sizeof(uint8_t), hipMemcpyHostToDevice));
	CHK(hipMemset(d_cost_volume, 255.0, volume_size * sizeof(__half)));//previously float

	// Copy the params into the gpu
	float* h_params_ref = get_params_cam(ref, 1, cam_vector);
	float* h_params_cam = get_params_cam(ref, 0, cam_vector);

#ifdef DEBUG // print the params
	printf("Print params :\n");
	for (int i = 0; i < 39; ++i) {
		printf("%f ", h_params_ref[i]);
	}
	printf("\n");
	for (int i = 0; i < 39; ++i) {
		printf("%f ", h_params_cam1[i]);
	}
	printf("\n");
#endif
	//Used in naive
	//CHK(hipMalloc((void**) &d_param_ref, sizeof(double) * 39));
	//CHK(hipMalloc((void**) &d_param_cam1, sizeof(double) * 39));
	//CHK(hipMalloc((void**) &d_param_cam2, sizeof(double) * 39));
	//CHK(hipMalloc((void**) &d_param_cam3, sizeof(double) * 39));
	//CHK(hipMemcpy(d_param_ref, h_params_ref, sizeof(double) * 39, hipMemcpyHostToDevice)); 
	//CHK(hipMemcpy(d_param_cam1, h_params_cam1, sizeof(double) * 39, hipMemcpyHostToDevice));
	//CHK(hipMemcpy(d_param_cam2, h_params_cam2, sizeof(double) * 39, hipMemcpyHostToDevice));
	//CHK(hipMemcpy(d_param_cam3, h_params_cam3, sizeof(double) * 39, hipMemcpyHostToDevice));
	//Used in coalesced
	CHK(hipMemcpyToSymbol(HIP_SYMBOL(d_param_ref), h_params_ref, sizeof(float) * 21));
	CHK(hipMemcpyToSymbol(HIP_SYMBOL(d_param_cam), h_params_cam, sizeof(float) * 54));

	// Define the kernel launch parameters
	dim3 block_size(BLOCKSIZE, BLOCKSIZE); //Number of threads per block (size of blocks) 16*16=256 or 32*32=1024
	dim3 grid_size((width + BLOCKSIZE-1) / BLOCKSIZE, (height + BLOCKSIZE-1) / BLOCKSIZE, z_planes); //Assure that all pixels are covered with (width+15)/16 and (height+15)/16
	printf("Launching kernel with grid size: %d %d %d\n", grid_size.x, grid_size.y, grid_size.z);
	printf("Launching kernel with block size: %d %d\n", block_size.x, block_size.y);
	// Parameters for shared memory
	int pad = window / 2;
	int shared_mem_size = (BLOCKSIZE + 2 * pad) * (BLOCKSIZE + 2 * pad) * sizeof(uint8_t);

	// launch 1 kernel per camera
	start = start_cuda_timer();
	/*naive_sweeping_plane_kernel <<<grid_size, block_size>>> (
		d_im_ref, d_im_cam1, d_param_ref, d_param_cam1, d_cost_volume, width, height, z_planes, window);
	naive_sweeping_plane_kernel <<<grid_size, block_size>>> (
		d_im_ref, d_im_cam2, d_param_ref, d_param_cam2, d_cost_volume, width, height, z_planes, window);
	naive_sweeping_plane_kernel <<<grid_size, block_size>>> (
		d_im_ref, d_im_cam3, d_param_ref, d_param_cam3, d_cost_volume, width, height, z_planes, window);*/
	//end_cuda_timer(start, "Naive GPU");
	/*params_sweeping_plane_kernel <<<grid_size, block_size>>> (
		d_im_ref, d_im_cam1, d_cost_volume,0, width, height, z_planes, window);
	params_sweeping_plane_kernel <<<grid_size, block_size>>> (
		d_im_ref, d_im_cam2, d_cost_volume,1, width, height, z_planes, window);
	params_sweeping_plane_kernel <<<grid_size, block_size>>> (
		d_im_ref, d_im_cam3, d_cost_volume,2, width, height, z_planes, window);
	end_cuda_timer(start, "Params optimized GPU");*/
	shared_sweeping_plane_kernel <<<grid_size, block_size, shared_mem_size>>> (
		d_im_ref, d_im_cam1, d_cost_volume, 0, width, height, z_planes, window);
	shared_sweeping_plane_kernel <<<grid_size, block_size, shared_mem_size>>> (
		d_im_ref, d_im_cam2, d_cost_volume, 1, width, height, z_planes, window);
	shared_sweeping_plane_kernel <<<grid_size, block_size, shared_mem_size>>> (
		d_im_ref, d_im_cam3, d_cost_volume, 2, width, height, z_planes, window);
	end_cuda_timer(start, "Shared GPU");
	CHK(hipGetLastError());
	CHK(hipDeviceSynchronize());
	CHK(hipMemcpy(h_cost_volume, d_cost_volume, sizeof(__half) * volume_size, hipMemcpyDeviceToHost));

Error:
	CHK(hipFree(d_im_ref));
	CHK(hipFree(d_im_cam1));
	CHK(hipFree(d_im_cam2));
	CHK(hipFree(d_im_cam3));
	CHK(hipFree(d_cost_volume));
	//Used in naive
	//CHK(hipFree(d_param_ref));
	//CHK(hipFree(d_param_cam));
	//CHK(hipFree(d_param_cam1));
	//CHK(hipFree(d_param_cam2));
	//CHK(hipFree(d_param_cam3));
	delete[] ref_flattened;
	delete[] cam1_flattened;
	delete[] cam2_flattened;
	delete[] cam3_flattened;

	// Needed for profiling
	CHK(hipDeviceReset());
}